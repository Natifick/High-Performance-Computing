#include "hip/hip_runtime.h"
#include <array>
#include <iostream>
#include <stdio.h>
#include <png.h>

#include "utils.cuh"

// ----- Read-write PNG -----

void png_to_arr(png_bytepp& row_pointers, int** array,
                const int& n_rows, const int& n_cols) {
    *array = (int*)malloc(sizeof(int) * n_cols * n_rows);
    for (unsigned int y = 0; y < n_rows; y++) {
        png_bytep row = row_pointers[y];
        for (unsigned int x = 0; x < n_cols; x++) {
            png_bytep px = &(row[x * 4]);
            // Grayscale on go
            (*array)[y*n_cols + x] = (int) (px[3] / 255) * (px[0] + px[1] + px[2]) / 3;
            /*printf("%4d, %4d = RGBA(%3d, %3d, %3d, %3d)\n", x, y, px[0], px[1], px[2], px[3]);*/
            //png_byte old[4 * sizeof(png_byte)];
            //memcpy(old, px, sizeof(old));
            
            //px[0] = 255 - old[0];
            //px[1] = 255 - old[1];
            //px[2] = 255 - old[2];
        }
    }
}

void arr_to_png(png_bytepp& row_pointers, int** array,
                const int& n_rows, const int& n_cols) {
    for (unsigned int y = 0; y < n_rows; y++) {
        png_bytep row = row_pointers[y];
        for (unsigned int x = 0; x < n_cols; x++) {
            png_bytep px = &(row[x * 4]);
            px[0] = px[1] = px[2] = (*array)[y*n_cols + x];
            // Grayscale on go
        }
    }
}

void read_png(std::string file_name, int& n_rows, int& n_cols, 
              png_infop& info_ptr, png_bytepp& row_pointers) {
    FILE *fp = fopen(file_name.c_str(), "rb");
    png_structp png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    info_ptr = png_create_info_struct(png_ptr);  
    png_init_io(png_ptr, fp);
    png_read_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    n_cols = png_get_image_width(png_ptr, info_ptr);
    n_rows = png_get_image_height(png_ptr, info_ptr);
    row_pointers = png_get_rows(png_ptr, info_ptr);
    png_destroy_read_struct(&png_ptr, NULL, NULL); 
    fclose(fp);
}

void write_png(std::string file_name, png_infop& info_ptr, png_bytepp& row_pointers) {
    FILE *fp = fopen(file_name.c_str(), "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_init_io(png_ptr, fp);
    png_set_rows(png_ptr, info_ptr, row_pointers);
    png_write_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}

// ----- GPU code -----
// Check all errors that occur
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void zeroKernel(float* array, int size) {
    int gid = getGid();
    if (gid < size) {
        array[gid] = 0;
    }
}

__global__ void stencilKernel(float* inArray, float* outArray, int size) {
    int gid = getGid();
    if (gid < size) {
        outArray[gid] = inArray[gid] * inArray[gid];
    }
}

__global__ void adderKernel(float* array, int size) {
    int gid = getGid();
    if (gid < size) {
        //array[0] = array[0] + 1;
        atomicAdd(&array[0], 1);
    }
}

// CPU code
int main() {
    int n_rows, n_cols;
    // Some stuff needed by png-reader
    png_infop info_ptr;
    png_bytepp row_pointers;
    std::string in_filename = "image.png", out_filename = "out.png";
    // ----- Read data from png -----
    read_png(in_filename, n_rows, n_cols, info_ptr, row_pointers);
    int* array;
    png_to_arr(row_pointers, &array, n_rows, n_cols);
    
    // ----- Process the data -----
    
    // TODO

    // ----- Write data back to another png -----
    arr_to_png(row_pointers, &array, n_rows, n_cols);
    write_png(out_filename, info_ptr, row_pointers);
    
    return 0;
}
